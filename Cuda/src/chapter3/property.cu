// 3.3 Querying Devices
#include <cstdio>

#include "common.h"

using namespace std;

// Parameters of devices, see more in "CUDA Programming Guide".
int main(int argc, char **argv) {

    printf("%s starting...\n", argv[0]);

    // Find cuda device.
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    if (error_id != hipSuccess) {
        printf("hipGetDeviceCount returned %d\n->%s\n", (int)error_id, hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }

    if (deviceCount == 0) {
        printf("There are no available device(s) that support CUDA.\n");
    }
    else {
        printf("Detectd %d CUDA Capable device(s).\n", deviceCount);
    }

    int dev = 0, driverVersion = 0, runtimeVersion = 0;
    hipSetDevice(dev);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Device %d: \"%s\".\n", dev, deviceProp.name);

    // Find driver.
    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);
    printf(
        "CUDA Driver Version / Runtime Version %d.%d / %d.%d\n",
        driverVersion / 1000, (driverVersion % 100) / 10, runtimeVersion / 1000, (runtimeVersion % 100) / 10
    );
    printf("CUDA Capability Major / Minor version number: %d.%d\n", deviceProp.major, deviceProp.minor);
    printf("Total amount of global memory: %0.2lf GBytes (%zu bytes)\n", (double)deviceProp.totalGlobalMem / (1073741824.0), deviceProp.totalGlobalMem);
    printf("GPU Clock rate: %.0f MHz (%0.2f GHz)\n", deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);
    printf("Memory Clock rate: %.0f Mhz\n", deviceProp.memoryClockRate * 1e-3f);
    printf("Memory Bus Width: %d-bit\n", deviceProp.memoryBusWidth);

    if (deviceProp.l2CacheSize) {
        printf("L2 Cache Size: %d bytes\n", deviceProp.l2CacheSize);
    }

    printf(
        "Max Texture Dimension Size (x, y, z) 1D = (%d), 2D = (%d, %d), 3D = (%d, %d, %d)\n",
        deviceProp.maxTexture1D, deviceProp.maxTexture2D[0], deviceProp.maxTexture2D[1],
        deviceProp.maxTexture3D[0], deviceProp.maxTexture3D[1], deviceProp.maxTexture3D[2]
    );
    printf(
        "Max Layered Texture Size (dim) x layers 1D = (%d) x %d, 2D = (%d, %d) x %d\n",
        deviceProp.maxTexture1DLayered[0], deviceProp.maxSurface1DLayered[1],
        deviceProp.maxTexture2DLayered[0], deviceProp.maxTexture2DLayered[1], deviceProp.maxTexture2DLayered[2]
    );

    printf("Total amount of constant memory: %zu bytes\n", deviceProp.totalConstMem);
    printf("Total amount of shared memory per block: %zu bytes\n", deviceProp.sharedMemPerBlock);
    printf("Total number of registers available per block: %d\n", deviceProp.regsPerBlock);
    printf("Total number of multiprocessors: %d\n", deviceProp.multiProcessorCount);
    printf("Warp size: %d\n", deviceProp.warpSize);
    printf("Maximum number of threads per multiprocessor: %d\n", deviceProp.maxThreadsPerMultiProcessor);
    printf("Maximum number of threads per block: %d\n", deviceProp.maxThreadsPerBlock);
    printf("Maximum sizes of each dimension of a block: %d x %d x %d\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
    printf("Maximum sizes of each dimension of a grid: %d x %d x %d\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
    printf("Maximum memory pitch: %zu bytes\n", deviceProp.memPitch);

    return 0;
}

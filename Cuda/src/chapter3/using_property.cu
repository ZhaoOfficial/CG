// 3.4 Using Device Properties
#include <stdio.h>

#include "common.h"

int main() {
    int device;
    HANDLE_ERROR(hipGetDevice(&device));
    printf("ID of current CUDA device: %d\n", device);

    // fill a `hipDeviceProp_t` struct with
    // the properties we need our device have
    hipDeviceProp_t prop;
    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = 7;
    prop.minor = 5;

    // then we can choose the device
    HANDLE_ERROR(hipChooseDevice(&device, &prop));
    printf("ID of CUDA device closest to 7.5: %d\n", device);
    HANDLE_ERROR(hipSetDevice(device));

    return 0;
}

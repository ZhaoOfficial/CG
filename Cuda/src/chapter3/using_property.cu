// 3.4 Using Device Properties
#include <cstdio>

#include "common.h"

int main() {
    int device;
    HANDLE_ERROR(hipGetDevice(&device));
    std::printf("ID of current CUDA device: %d\n", device);

    // Fill a `hipDeviceProp_t` struct with
    // the properties we need our device have.
    hipDeviceProp_t prop;
    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = 7;
    prop.minor = 5;

    // Then we can choose the device.
    HANDLE_ERROR(hipChooseDevice(&device, &prop));
    std::printf("ID of CUDA device closest to 7.5: %d\n", device);
    HANDLE_ERROR(hipSetDevice(device));

    return 0;
}

#include "hip/hip_runtime.h"
// 3.2.3 Passing Parameters
#include "common.h"

__global__ void addKernel(int a, int b, int *c) {
    *c = a + b;
}

int main() {
    int c;
    int *dev_c;

    HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(int)));
    addKernel<<<1, 1>>>(2, 7, dev_c);
    HANDLE_ERROR(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));
    std::printf("2 + 7 = %d\n", c);

    HANDLE_ERROR(hipFree(dev_c));
    return 0;
}

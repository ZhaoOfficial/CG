#include "hip/hip_runtime.h"
// 3.2.3 Passing Parameters
#include <stdio.h>

#include "common.h"

__global__ void addKernel(int a, int b, int *c) {
    *c = a + b;
}

int main() {
    int c;
    int *dev_c;

    HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(int)));
    addKernel<<<1, 1>>>(2, 7, dev_c);
    HANDLE_ERROR(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));
    printf("2 + 7 = %d\n", c);

    hipFree(dev_c);
    return 0;
}

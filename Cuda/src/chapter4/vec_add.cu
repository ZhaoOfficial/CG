#include "hip/hip_runtime.h"
// 4.2.1 summing Vectors
#include <stdio.h>

#include "common.h"

constexpr int N = 10;

__global__ void addKernel(int const *a, int const *b, int *c) {
    int tid = threadIdx.x;
    if (tid < N) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;
    
    // allocate memory on the GPU
    HANDLE_ERROR(hipMalloc((void**)&dev_a, sizeof(int) * N));
    HANDLE_ERROR(hipMalloc((void**)&dev_b, sizeof(int) * N));
    HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(int) * N));

    // fill the arrays `a` and `b` on the CPU;
    for (int i{}; i < N; ++i) {
        a[i] = -i;
        b[i] = i * i;
    }

    // copy the arrays `a` and `b` to the GPU
    HANDLE_ERROR(hipMemcpy(dev_a, a, sizeof(int) * N, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, sizeof(int) * N, hipMemcpyHostToDevice));

    addKernel<<<1, N>>>(dev_a, dev_b, dev_c);

    // copy the array `c` back from the GPU to the CPU
    HANDLE_ERROR(hipMemcpy(c, dev_c, sizeof(int) * N, hipMemcpyDeviceToHost));

    // display the results
    for (int i{}; i < N; ++i) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    return 0;
}

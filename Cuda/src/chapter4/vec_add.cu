#include "hip/hip_runtime.h"
// 4.2.1 summing Vectors
#include <cstdio>

#include "common.h"

constexpr int N = 10;

__global__ void addKernel(int const *a, int const *b, int *c) {
    int tid = threadIdx.x;
    if (tid < N) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;
    
    // Allocate memory on the GPU.
    HANDLE_ERROR(hipMalloc((void**)&dev_a, sizeof(int) * N));
    HANDLE_ERROR(hipMalloc((void**)&dev_b, sizeof(int) * N));
    HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(int) * N));

    // Fill the arrays `a` and `b` on the CPU.
    for (int i{}; i < N; ++i) {
        a[i] = -i;
        b[i] = i * i;
    }

    // Copy the arrays `a` and `b` to the GPU.
    HANDLE_ERROR(hipMemcpy(dev_a, a, sizeof(int) * N, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, sizeof(int) * N, hipMemcpyHostToDevice));

    addKernel<<<1, N>>>(dev_a, dev_b, dev_c);

    // Copy the array `c` back from the GPU to the CPU.
    HANDLE_ERROR(hipMemcpy(c, dev_c, sizeof(int) * N, hipMemcpyDeviceToHost));

    // Display the results.
    for (int i{}; i < N; ++i) {
        std::printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipFree(dev_b));
    HANDLE_ERROR(hipFree(dev_c));
    return 0;
}

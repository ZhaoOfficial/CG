#include "hip/hip_runtime.h"
// 5.2.2 GPU Ripple Using Threads
#include <cmath>
#include <cstdio>
#include <filesystem>
#include <string>
#include <vector>

#include "common.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

constexpr int DIM = 1024;

template <typename T = float>
__global__ void kernel(uint8_t *ptr, int tick, unsigned int x_dim, unsigned int y_dim) {
    for (unsigned int y = blockIdx.y * blockDim.y + threadIdx.y; y < y_dim; y += gridDim.y * blockDim.y) {
        for (unsigned int x = blockIdx.x * blockDim.x + threadIdx.x; x < x_dim; x += gridDim.x * blockDim.x) {
            unsigned int pixel_id = (x + y * x_dim) * 4;
            int rx = x - x_dim / 2;
            int ry = y - y_dim / 2;
            T r = std::sqrt(T(rx * rx + ry * ry));
            uint8_t value = (
                T(128.0) + T(127.0) * std::cos(
                    (r / T(15.0) - tick / T(10.0)) * pi<T>
                ) / (r / T(15.0) + T(1.0))
            );

            ptr[pixel_id + 0] = value;
            ptr[pixel_id + 1] = value;
            ptr[pixel_id + 2] = value;
            ptr[pixel_id + 3] = 255;
        }
    }
}

int main(int argc, char **argv) {
    if (argc != 2) {
        std::printf("Output path invalid.\n");
        return EXIT_FAILURE;
    }
    std::filesystem::path file_path(argv[1]);
    if (file_path.has_extension()) {
        std::printf("Output path [%s] invalid.\n", file_path.string().c_str());
        return EXIT_FAILURE;
    }
    std::printf("Output path: %s\n", argv[1]);

    std::vector<uint8_t> bitmap(DIM * DIM * 4);
    uint8_t *dev_bitmap;

    HANDLE_ERROR(hipMalloc((void**)&dev_bitmap, bitmap.size() * sizeof(uint8_t)));

    dim3 block_size(16, 16);
    dim3 grid_size(32, 32);
    for (int tick{}; tick < 40; ++tick) {
        kernel<float><<<grid_size, block_size>>>(dev_bitmap, tick, DIM, DIM);

        HANDLE_ERROR(hipMemcpy(bitmap.data(), dev_bitmap, bitmap.size() * sizeof(uint8_t), hipMemcpyDeviceToHost));
        std::filesystem::path out_path = file_path / (std::to_string(tick) + ".png");
        stbi_flip_vertically_on_write(1);
        stbi_write_png(out_path.string().c_str(), DIM, DIM, 4, bitmap.data(), 0);
        std::printf("%s output successfully!\n", out_path.string().c_str());
    }
    HANDLE_ERROR(hipFree(dev_bitmap));
    return 0;
}

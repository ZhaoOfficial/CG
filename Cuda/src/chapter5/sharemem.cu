#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstdio>
#include <numeric>

#include "common.h"

constexpr int ThreadsPerBlock{256};

__global__ void dotProduct(int const N, float const* a, float const* b, float *c) {
    __shared__ float cache[ThreadsPerBlock];

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cache_index = threadIdx.x;

    float result{};
    for (; tid < N; tid += blockDim.x * gridDim.x) {
        result += a[tid] * b[tid];
    }

    // Store the thread result in the cache.
    cache[cache_index] = result;
    __syncthreads();

    // Reduction process, reduce the result as one.
    // This requires blockDim.x to be power of 2.
    for (int i = blockDim.x / 2; i != 0; i /= 2) {
        if (cache_index < i) {
            cache[cache_index] += cache[cache_index + i];
        }
        __syncthreads();
    }
    if (cache_index == 0) {
        c[blockIdx.x] = cache[0];
    }
    // Warp reduce
    if (threadIdx.x < 32) {
        // Need volatile to implicit synchronization
        volatile float* vss = cache;
        float val = vss[threadIdx.x];
        val += __shfl_down_sync(0xffffffff, val, 16);
        val += __shfl_down_sync(0xffffffff, val, 8);
        val += __shfl_down_sync(0xffffffff, val, 4);
        val += __shfl_down_sync(0xffffffff, val, 2);
        val += __shfl_down_sync(0xffffffff, val, 1);
        // Store the block result in `c`.
        if (cache_index == 0) {
            c[blockIdx.x] = val;
        }
    }
}

int main(int argc, char **argv) {

    constexpr int ArraySize{33 * 1024};
    constexpr int BlocksPerGrid{std::min(32, (ArraySize + ThreadsPerBlock - 1) / ThreadsPerBlock)};

    float *a = new float[ArraySize];
    float *b = new float[ArraySize];
    float *c = new float[BlocksPerGrid];
    float *dev_a{}, *dev_b{}, *dev_c{};

    HANDLE_ERROR(hipMalloc((void**)&dev_a, ArraySize * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b, ArraySize * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_c, BlocksPerGrid * sizeof(int)));

    for (int i{}; i < ArraySize; ++i) {
        a[i] = i;
        b[i] = i * 2;
    }

    HANDLE_ERROR(hipMemcpy(dev_a, a, ArraySize * sizeof(int), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, ArraySize * sizeof(int), hipMemcpyHostToDevice));

    dotProduct<<<BlocksPerGrid, ThreadsPerBlock>>>(ArraySize, dev_a, dev_b, dev_c);

    HANDLE_ERROR(hipMemcpy(c, dev_c, BlocksPerGrid * sizeof(int), hipMemcpyDeviceToHost));

    float result = std::accumulate(c, c + BlocksPerGrid, 0.0f);
    std::printf("Result = %f\n", result);
    auto sumOfSquare = [](float x) -> float { return x * (x + 1) * (2 * x + 1) / 6; };
    std::printf("Expected result = %f\n", sumOfSquare(ArraySize - 1) * 2.0f);

    delete[] a;
    delete[] b;
    delete[] c;
    return 0;
}

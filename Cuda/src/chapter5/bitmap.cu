#include "hip/hip_runtime.h"
// 5.3.2 shared memory bitmap
#include <cmath>
#include <cstdint>
#include <cstdio>
#include <filesystem>
#include <string>
#include <vector>

#include "common.h"

constexpr int DIM = 1024;

template <typename T = float>
__global__ void kernel(uint8_t *ptr, int x_dim, int y_dim) {
    // 2d share memory
    __shared__ float cache[16][16];
    constexpr T period = T(128.0);

    for (int y = blockIdx.y * blockDim.y + threadIdx.y; y < y_dim; y += gridDim.y * blockDim.y) {
        for (int x = blockIdx.x * blockDim.x + threadIdx.x; x < x_dim; x += gridDim.x * blockDim.x) {
            int pixel_id = (x + y * x_dim) * 4;

            cache[threadIdx.x][threadIdx.y] = T(255.0 / 4.0) * (
                std::sin(x * T(2.0) * pi<T> / period) + T(1.0)
            ) * (
                std::sin(y * T(2.0) * pi<T> / period) + T(1.0)
            );

            // Synchronization before any thread communication.
            __syncthreads();

            ptr[pixel_id + 0] = 0;
            ptr[pixel_id + 1] = cache[15 - threadIdx.x][15 - threadIdx.y];
            ptr[pixel_id + 2] = 0;
            ptr[pixel_id + 3] = 255;
        }
    }
}

int main(int argc, char **argv) {
    PathChecker::checkFilePath(argc, argv, ".png");
    Bitmap bitmap(DIM, DIM);
    dim3 block_size(16, 16);
    dim3 grid_size(DIM / 16, DIM / 16);

    kernel<float><<<grid_size, block_size>>>(bitmap.dev_bitmap, DIM, DIM);
    bitmap.memcpyDeviceToHost();
    bitmap.toImage(argv[1]);

    return 0;
}

